#include "hip/hip_runtime.h"
﻿#include "CudaProcess.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
using namespace std;

__global__ void kernel(int* pSrc1, int* pSrc2, int* pResult, int length)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= length) {
		return;
	}

	pResult[idx] = pSrc1[idx] + pSrc2[idx];
	return;
}
int main() {
	Process();
}

void Process() {
	int length = 1024;
	size_t size = sizeof(int) * length;

	int* pHostSrc1;
	int* pHostSrc2;
	int* pHostResult;

	int* pDevSrc1;
	int* pDevSrc2;
	int* pDevResult;

	hipHostMalloc(&pHostSrc1, size);
	hipHostMalloc(&pHostSrc2, size);
	hipHostMalloc(&pHostResult, size);

	hipMalloc(&pDevSrc1, size);
	hipMalloc(&pDevSrc2, size);
	hipMalloc(&pDevResult, size);

	//
	pHostSrc1[2] = 3;
	pHostSrc2[2] = 5;

	hipMemcpy(pDevSrc1, pHostSrc1, size, hipMemcpyHostToDevice);
	hipMemcpy(pDevSrc2, pHostSrc2, size, hipMemcpyHostToDevice);

	dim3 block(128, 1, 1);
	dim3 grid((length + 128 - 1) / 128, 1, 1);
	kernel << <grid, block >> > (pDevSrc1, pDevSrc2, pDevResult, length);
	hipMemcpy(pHostResult, pDevResult, size, hipMemcpyDeviceToHost);


	cout << pHostSrc1[2] << ":" << pHostSrc2[2] << ":" << pHostResult[2] << endl;
	cout << "end" << endl;

	hipFree(pDevSrc1);
	hipFree(pDevSrc2);
	hipFree(pDevResult);

	hipHostFree(pHostSrc1);
	hipHostFree(pHostSrc2);
	hipHostFree(pHostResult);

	hipDeviceReset();


}
