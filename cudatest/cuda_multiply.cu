#include "hip/hip_runtime.h"
﻿#include<stdio.h>

#include<hip/hip_runtime.h>
#include ""
#include <ctime>
#include <iostream>

using namespace std;

#define BLOCK 16
#define WIDTH 1024


float* d_A, * d_B, * d_C;
__global__ void d_multiply0(float* A, float* B, float* C);

__global__ void d_multiply0(float* A, float* B, float* C) {
	unsigned int r = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int c = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int i;
	float tmp;
	tmp = 0.0f;
	for (i = 0; i < WIDTH; i++) {
		tmp += A[WIDTH * r * i] * B[WIDTH * i + c];
	}
	C[WIDTH * r + c] = tmp;

}
void d_multiply(float* A, float* B, float* C) {

	clock_t start = clock();
	hipMemcpy(d_A, A, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);

	dim3 grid(WIDTH / BLOCK, WIDTH / BLOCK);
	dim3 block(BLOCK, BLOCK);
	d_multiply0 << < grid, block >> > (A, B, C);
	hipMemcpy(C, d_C, sizeof(float) * WIDTH * WIDTH, hipMemcpyDeviceToHost);
	clock_t end = clock();
	cout << "CUDA計算時間:" << (double)(end - start) << endl;
}
float ha_A[WIDTH * WIDTH];
float ha_B[WIDTH * WIDTH];
float ha_C[WIDTH * WIDTH];
void d_process() {



	unsigned int i;

	hipMalloc((void**)&d_A, sizeof(float) * WIDTH * WIDTH);
	hipMalloc((void**)&d_B, sizeof(float) * WIDTH * WIDTH);
	hipMalloc((void**)&d_C, sizeof(float) * WIDTH * WIDTH);

	for (i = 0; i < WIDTH * WIDTH; i++) {
		ha_A[i] = (float)i;
		ha_B[i] = (float)i;
	}
	d_multiply(ha_A, ha_B, ha_C);
}