#include "hip/hip_runtime.h"
﻿#include<stdio.h>

#include<hip/hip_runtime.h>
#include ""
#include <ctime>
#include <iostream>

using namespace std;

#define BLOCK 16
#define WIDTH 1024


float* d_A, * d_B, * d_C;

__global__ void d_multiply0(float* A, float* B, float* C) {
	unsigned int r = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int c = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int i;
	float tmp;
	tmp = 0.0f;
	for (i = 0; i < WIDTH; i++) {
		tmp += A[WIDTH * r + i] * B[WIDTH * i + c];
	}
	C[WIDTH * r + c] = tmp;

}
float ha_A[WIDTH * WIDTH];
float ha_B[WIDTH * WIDTH];
float ha_C[WIDTH * WIDTH];
void d_multiply() {

	clock_t start = clock();
	size_t size = sizeof(float) * WIDTH * WIDTH;
	hipMemcpy(d_A, ha_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, ha_B, size, hipMemcpyHostToDevice);

	dim3 grid(WIDTH / BLOCK, WIDTH / BLOCK);
	dim3 block(BLOCK, BLOCK);
	d_multiply0 << < grid, block >> > (d_A, d_B, d_C);
	hipMemcpy(ha_C, d_C, size, hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	clock_t end = clock();
	cout << "CUDA計算時間:" << (double)(end - start) << endl;
}

float* d_process() {



	unsigned int i;

	hipMalloc((void**)&d_A, sizeof(float) * WIDTH * WIDTH);
	hipMalloc((void**)&d_B, sizeof(float) * WIDTH * WIDTH);
	hipMalloc((void**)&d_C, sizeof(float) * WIDTH * WIDTH);

	for (i = 0; i < WIDTH * WIDTH; i++) {
		ha_A[i] = (float)i;
		ha_B[i] = (float)i;
	}
	d_multiply();
	return &ha_C[0];
}