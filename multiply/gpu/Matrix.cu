#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <timer.h>

#define BLOCK 16
#define WIDTH 1024

float h_A[WIDTH * WIDTH];
float h_B[WIDTH * WIDTH];
float h_C[WIDTH * WIDTH];

float *d_A, *d_B, *d_C;

void h_multiply(float *A, float *B, float *C);
__global__ void d_multiply(float *A, float *B, float *C);

int main() {
  unsigned int i;

  hipMalloc((void**)&d_A, sizeof(float) * WIDTH * WIDTH);
  hipMalloc((void**)&d_B, sizeof(float) * WIDTH * WIDTH);
  hipMalloc((void**)&d_C, sizeof(float) * WIDTH * WIDTH);

  for (i = 0; i < (WIDTH * WIDTH); i++) {
    h_A[i] = (float)i;
    h_B[i] = (float)i;
  }

  StartTimer();

  hipMemcpy(d_A, h_A, sizeof(float) * WIDTH * WIDTH), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, sizeof(float) * WIDTH * WIDTH), hipMemcpyHostToDevice);

  dim3 grid(WIDTH/BLOCK, WIDTH/BLOCK);
  dim3 block(BLOCK, BLOCK);

  d_multiply0 <<<grid,block >>>(d_A,d_B,d_C);

  hipMemcpy(h_B, d_B, sizeof(float) * WIDTH * WIDTH), hipMemcpyDeviceToHost);

  printf("デバイス計算時間: %f(ms)", GetTimer());
  printf("デバイス計算結果: %f\n", h_C[WIDTH*WIDTH-1]);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);



//
  StartTimer();
  h_multiply(h_A, h_B, h_C);
  printf("ホスト計算時間：%f\n", );
  printf("ホスト計算結果：%f\n", h_C[WIDTH * WIDTH - 1]);
}
void h_multiply(float *A, float *B, float *C) {
  unsigned int r, c, i;
  float tmp;
  for (int r = 0; r < WIDTH; r++) {
    for (int c = 0; c < WIDTH; c++) {
      tmp = 0.0;
      for (int i = 0; i < WIDTH; i++) {
        tmp += A[WIDTH * r + i] * B[WIDTH * i + c];
      }
      C[WIDTH * r + c] = tmp;
    }
  }
}

__global__ void d_multiply0(float *A, float *B, float *C){
  unsigned int r= blockDim.y * blockIdx.y + threadIdx.y;
  unsigned int c= blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int i;
  flaot tmp;
  tmp =0.0f;
  for(i=0; i <WIDTH; i++){
    tmp +=A[WIDTH * r + i] * B[WIDTH * i + c];
  }
  C[WIDTH * r + c] = tmp;

}

__global__ void d_multiply1(float *A, float *B, float *C){
  unsigned int r= blockDim.y * blockIdx.y + threadIdx.y;
  unsigned int c= blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int i,j;

  float tmp;
  __shared__ float s_A[BLOCK][BLOCK];
  __shared__ float s_B[BLOCK][BLOCK];
  tmp = 0.0f;

  for (i =0; i< WIDTH, i += BLOCK){
    s_A[thredIdx.y][threadIdx.x] = A[WIDTH * r + i + threadIdx.x];
    s_B[thredIdx.y][threadIdx.x] = B[WIDTH * (i + threadIdx.y) + c ];
    __syncthreads();
    for(j=0; j < BLOCK; j++){
    tmp+= s_A[thredIdx.y][j] * s_B[j][threadIdx.x];
    __syncthreads();
  }
  c[WIDTH * r + c] = tmp;


}