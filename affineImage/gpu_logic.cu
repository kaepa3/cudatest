#include "hip/hip_runtime.h"
﻿#include <stdio.h>

#include<hip/hip_runtime.h>
#include ""
#include "HalconCpp.h"

using namespace HalconCpp;
using namespace std;


unsigned char* d_src, * d_dst;
double* d_matrix;

__global__ void  d_affine(unsigned char* src, double* matrix, int width, int height, int dstWidth, int dstHeight, unsigned char* dst) {

	unsigned int r = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int c = blockDim.x * blockIdx.x + threadIdx.x;

	double refX = matrix[0] * (c - ((double)dstWidth / 2)) +
		matrix[1] * (r - ((double)dstHeight / 2)) +
		((double)width / 2);
	double refY = matrix[3] * (c - ((double)dstWidth / 2)) +
		matrix[4] * (r - ((double)dstHeight / 2)) +
		((double)height / 2);
	int rX = (int)(refX + 0.5);
	int rY = (int)(refY + 0.5);
	unsigned char val = 0;
	if (rX >= 0 && rY >= 0 && rX < width && rY < height) {

		int idxRef = rY * width + rX;
		val = src[idxRef];
	}
	int idx = r * dstWidth + c;
	dst[idx] = val;
	return;
}

#define BLOCK 4
void gpu_affine() {

	HObject image;
	HalconCpp::ReadImage(&image, "sample.jpg");
	HObject imgR, imgG, imgB;
	Decompose3(image, &imgR, &imgG, &imgB);
	HObject gray;
	Rgb3ToGray(imgR, imgG, imgB, &gray);

	HTuple p, tp, w, h;
	GetImagePointer1(gray, &p, &tp, &w, &h);

	HTuple hommat2dIdentity;
	HomMat2dIdentity(&hommat2dIdentity);

	HTuple hommat2dscale;
	HomMat2dScale(hommat2dIdentity, 2, 2, 0, 0, &hommat2dscale);

	HTuple hommat2dinvert;
	HomMat2dInvert(hommat2dscale, &hommat2dinvert);

	HTuple hommat2drotate;
	double rad = (45.0 / 180.0) * PI;
	HomMat2dRotate(hommat2dinvert, rad, h / 2, w / 2, &hommat2drotate);

	unsigned char* ptr = (unsigned char*)p.L();
	int length = w.I() * h.I();
	int length_2 = w.I() * 2 * h.I() * 2;
	size_t matrixSize = sizeof(double) * 6;
	unsigned char* buffer = new unsigned char[length_2];

	hipMalloc((void**)&d_src, length);
	hipMalloc((void**)&d_dst, length_2);
	hipMalloc((void**)&d_matrix, matrixSize);

	clock_t  start = clock();

	hipMemcpy(d_src, ptr, length, hipMemcpyHostToDevice);
	hipMemcpy(d_dst, buffer, length_2, hipMemcpyHostToDevice);
	hipMemcpy(d_matrix, hommat2drotate.ToDArr(), matrixSize, hipMemcpyHostToDevice);

	int width = w.I() * 2;
	dim3 grid(900 / BLOCK, 900/ BLOCK);
	dim3 block(BLOCK, BLOCK);
	d_affine << < grid, block >> > (d_src, d_matrix , w.I(), h.I(), w.I() * 2, h.I() * 2, d_dst);

	hipMemcpy(buffer, d_dst, length_2, hipMemcpyDeviceToHost);
	clock_t  end = clock();
	cout << end - start << ":" << CLOCKS_PER_SEC << endl;
	HImage dst;
	GenImage1(&dst, "byte", w.I() * 2, h.I() * 2, (Hlong)buffer);
	delete[] buffer;
	WriteImage(dst, "tiff", 0, "gpu_affine");

}
