#include "hip/hip_runtime.h"
﻿#include <stdio.h>

#include<hip/hip_runtime.h>
#include ""
#include "HalconCpp.h"
#include "common_val.h"

using namespace HalconCpp;
using namespace std;


unsigned char* d_src, * d_dst;
double* d_matrix;

__global__ void  d_affine(unsigned char* src, double* matrix, int width, int height, int dstWidth, int dstHeight, unsigned char* dst) {

	unsigned int r = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int c = blockDim.x * blockIdx.x + threadIdx.x;
	if ((c < dstWidth) && (r < dstHeight)) {

		double refX = matrix[0] * (c - ((double)dstWidth / 2)) +
			matrix[1] * (r - ((double)dstHeight / 2)) +
			((double)width / 2);
		double refY = matrix[3] * (c - ((double)dstWidth / 2)) +
			matrix[4] * (r - ((double)dstHeight / 2)) +
			((double)height / 2);
		int rX = (int)(refX + 0.5);
		int rY = (int)(refY + 0.5);
		unsigned char val = 0;
		if (rX >= 0 && rY >= 0 && rX < width && rY < height) {

			int idxRef = rY * width + rX;
			val = src[idxRef];
		}
		int idx = (r * dstWidth) + c;
		if (idx < dstWidth * dstHeight)
		{
			dst[idx] = val;
		}

	}
	return;
}

#define BLOCKWIDTH 	64
#define BLOCKHEIGHT 16
void gpu_affine() {

	HObject image;
	HalconCpp::ReadImage(&image, READIMAGE);
	HObject imgR, imgG, imgB;
	Decompose3(image, &imgR, &imgG, &imgB);
	HObject gray;
	Rgb3ToGray(imgR, imgG, imgB, &gray);

	HTuple p, tp, w, h;
	GetImagePointer1(gray, &p, &tp, &w, &h);

	HTuple hommat2dIdentity;
	HomMat2dIdentity(&hommat2dIdentity);

	HTuple hommat2dscale;
	HomMat2dScale(hommat2dIdentity, 2, 2, 0, 0, &hommat2dscale);

	HTuple hommat2dinvert;
	HomMat2dInvert(hommat2dscale, &hommat2dinvert);

	HTuple hommat2drotate;
	double rad = (45.0 / 180.0) * PI;
	HomMat2dRotate(hommat2dinvert, rad, 0, 0, &hommat2drotate);

	unsigned char* ptr = (unsigned char*)p.L();
	int length = w.I() * h.I();
	int length_2 = w.I() * 2 * h.I() * 2;
	size_t matrixSize = sizeof(double) * 6;
	unsigned char* buffer = new unsigned char[length_2];

	hipMalloc((void**)&d_src, length);
	hipMalloc((void**)&d_dst, length_2);
	hipMalloc((void**)&d_matrix, matrixSize);

	clock_t  start = clock();

	hipMemcpy(d_src, ptr, length, hipMemcpyHostToDevice);
	hipMemcpy(d_dst, buffer, length_2, hipMemcpyHostToDevice);
	hipMemcpy(d_matrix, hommat2drotate.ToDArr(), matrixSize, hipMemcpyHostToDevice);

	int width = w.I() * 2;
	int height = h.I() * 2;
	
	dim3 block(BLOCKWIDTH, BLOCKHEIGHT);
	dim3 grid((width + BLOCKWIDTH-1)/ BLOCKWIDTH, (height + BLOCKHEIGHT-1) / BLOCKHEIGHT);
	double s = HSystem::CountSeconds(); 
	d_affine << < grid, block >> > (d_src, d_matrix, w.I(), h.I(), w.I() * 2, h.I() * 2, d_dst);
	double e = HSystem::CountSeconds(); 
	cout << (e-s) * 1000 << ":only" << endl;


	hipMemcpy(buffer, d_dst, length_2, hipMemcpyDeviceToHost);
	hipFree(d_src);
	hipFree(d_dst);
	hipFree(d_matrix);
	clock_t  end = clock();
	cout << end - start << ":" << CLOCKS_PER_SEC << endl;
	HImage dst;
	GenImage1(&dst, "byte", w.I() * 2, h.I() * 2, (Hlong)buffer);
	WriteImage(dst, "tiff", 0, "gpu_affine");

	//for (int i = 0; i < width; i++)
	//{
	//	cout << "[" << +buffer[i] << "]";
	//}
	//cout << endl;

	delete[] buffer;
}
